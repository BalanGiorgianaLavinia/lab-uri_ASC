#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
		c[i] = a[i] + b[i];
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;
    const size_t block_size = 256;
    size_t num_blocks;
    hipError_t error;
    
    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1: Allocate the host's arrays
    host_array_a = (float*)malloc(N * sizeof(*host_array_a));
    if (host_array_a == NULL)
        exit(EXIT_FAILURE);

	host_array_b = (float*)malloc(N * sizeof(*host_array_b));
	if (host_array_b == NULL)
        exit(EXIT_FAILURE);

	host_array_c = (float*)malloc(N * sizeof(*host_array_c));
    if (host_array_c == NULL)
        exit(EXIT_FAILURE);
    
    // TODO 2: Allocate the device's arrays
    error = hipMalloc(&device_array_a, N * sizeof(*device_array_a));
    if (error != hipSuccess)
        exit(EXIT_FAILURE);

    error = hipMalloc(&device_array_b, N * sizeof(*device_array_b));
    if (error != hipSuccess)
        exit(EXIT_FAILURE);

    error = hipMalloc(&device_array_c, N * sizeof(*device_array_c));
	if (error != hipSuccess)
        exit(EXIT_FAILURE);


    // TODO 3: Check for allocation errors

    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.
    fill_array_float(host_array_a, N);
	fill_array_random(host_array_b, N);

    // TODO 5: Copy the host's arrays to device
    error = hipMemcpy(device_array_a, host_array_a, N * sizeof(*host_array_a), hipMemcpyHostToDevice);
	if (error != hipSuccess)
        exit(EXIT_FAILURE);


	error = hipMemcpy(device_array_b, host_array_b, N * sizeof(*host_array_b), hipMemcpyHostToDevice);
    if (error != hipSuccess)
        exit(EXIT_FAILURE);

    
    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).
    num_blocks = N / block_size;

	if (N % block_size)
		++num_blocks;

    add_arrays<<<num_blocks, block_size>>>(device_array_a,
                                        device_array_b,
		                                device_array_c, N);

    error = hipDeviceSynchronize();
    if (error != hipSuccess)
        exit(EXIT_FAILURE);

    
    // TODO 7: Copy back the results and then uncomment the checking function
    error = hipMemcpy(host_array_c,
                    device_array_c,
                    N * sizeof(*host_array_c),
                    hipMemcpyDeviceToHost);
    if (error != hipSuccess)
        exit(EXIT_FAILURE);

	check_task_2(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory
    free(host_array_a);
	free(host_array_b);
    free(host_array_c);
    
    hipFree(device_array_a);
    hipFree(device_array_b);
    hipFree(device_array_c);

    return 0;
}