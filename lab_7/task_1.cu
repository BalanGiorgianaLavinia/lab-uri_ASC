#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
		a[i] %= 2;
}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
		a[i] = blockIdx.x;
}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
		a[i] = threadIdx.x;
}

int main(void) {
    int nDevices;
    hipDeviceProp_t p;
    int *host_array;
    int *device_array;
    int n = 1 << 20;


    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

    // ~TODO 1~
    // For each device, show some details in the format below, 
    // then set as active device the first one (assuming there
    // is at least CUDA-capable device). Pay attention to the
    // type of the fields in the hipDeviceProp_t structure.
    //
    // Device number: <i>
    //      Device name: <name>
    //      Total memory: <mem>
    //      Memory Clock Rate (KHz): <mcr>
    //      Memory Bus Width (bits): <mbw>
    // 
    // Hint: look for hipGetDeviceProperties and hipSetDevice in
    // the Cuda Toolkit Documentation. 
    for (int i = 0; i < nDevices; ++i) {
        hipGetDeviceProperties(&p, i);
        printf("Device number: %d\n", i);
        printf("\tDevice name: %s\n", p.name);
        printf("\tTotal memory: %zu\n", p.totalGlobalMem);
        printf("\tMemory Clock Rate (KHz): %d\n", p.memoryClockRate);
        printf("\tMemory Bus Width (bits): %d\n", p.memoryBusWidth);
    }

    // ~TODO 2~
    // With information from example_2.cu, allocate an array with
    // integers (where a[i] = i). Then, modify the three kernels
    // above and execute them using 4 blocks, each with 4 threads.
    // Hint: num_elements = block_size * block_no (see example_2)
    //
    // You can use the fill_array_int(int *a, int n) function (from utils)
    // to fill your array as many times you want.

    host_array = (int *)malloc(n * sizeof(*host_array));

	hipMalloc(&device_array, n * sizeof(*device_array));

	fill_array_int(host_array, n);

	hipMemcpy(device_array, host_array, n * sizeof(*host_array), hipMemcpyHostToDevice);

    // ~TODO 3~
    // Execute kernel_parity_id kernel and then copy from 
    // the device to the host; call hipDeviceSynchronize()
    // after a kernel execution for safety purposes.
    //
    // Uncomment the line below to check your results
    kernel_parity_id<<<n / 4, 4>>>(device_array, n);

	hipDeviceSynchronize();

	hipMemcpy(host_array, device_array, n * sizeof(*host_array), hipMemcpyDeviceToHost);

    check_task_1(3, host_array);

    // ~TODO 4~
    // Execute kernel_block_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results
    kernel_block_id<<<n / 4, 4>>>(device_array, n);

	hipDeviceSynchronize();

	hipMemcpy(host_array, device_array, n * sizeof(*host_array), hipMemcpyDeviceToHost);

    check_task_1(4, host_array);

    // ~TODO 5~
    // Execute kernel_thread_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results
    kernel_thread_id<<<n / 4, 4>>>(device_array, n);

	hipDeviceSynchronize();

	hipMemcpy(host_array, device_array, n * sizeof(*host_array), hipMemcpyDeviceToHost);

    check_task_1(5, host_array);

    // TODO 6: Free the memory
    free(host_array);
    hipFree(device_array);
    

    return 0;
}
